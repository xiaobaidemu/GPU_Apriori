#ifndef TEST_VECTOR_AND_CU
#define TEST_VECTOR_AND_CU

#include "CUDA_header.cu"
#include "gpu_vectorAnd.cu"
#include "GPUMiner_Kernel.h"

void test_array_And_wS_woC()
{
	int in1Len = 2;
	int in2Len = in1Len*3;
	bytes_t* h_in1 = (bytes_t*)malloc( sizeof(bytes_t)*in1Len );
	bytes_t* h_in2 = (bytes_t*)malloc( sizeof(bytes_t)*in2Len );

	h_in1[0] = 2047;
	h_in1[1] = 65024;

	h_in2[0] = 65343;
	h_in2[1] = 32768;
	h_in2[2] = 0;
	h_in2[3] = 0;
	h_in2[4] = 65528;
	h_in2[5] = 256;	

	bytes_t* d_in1;
	bytes_t* d_in2;
	copyToGPU<bytes_t>( &d_in1, h_in1, in1Len );
	copyToGPU<bytes_t>( &d_in2, h_in2, in2Len );

	int listLen = 2;
	int* h_list = (int*)malloc( sizeof(int)*listLen );
	h_list[0] = 0;
	h_list[1] = 2;
	bytes_t* d_out;	
	int outLen = in1Len*listLen;
	GPUMALLOC( (void**)&d_out, sizeof(bytes_t)*outLen );
	
	arrayAndInit( listLen, in1Len );
	
	/*array_And_wS_woC( d_in1, in1Len, 
					   d_in2, h_list, listLen, 
					   d_out );*/
	table_t* d_countOut;
	GPUMALLOC( (void**)&d_countOut, sizeof(table_t)*outLen );
	array_And_woS_wC( d_in1, in1Len, 
					   d_in2, h_list, listLen, 
					   d_countOut ); 

	table_t* h_countOut;
	copyFromGPU( &h_countOut, d_countOut, outLen );
}


#endif