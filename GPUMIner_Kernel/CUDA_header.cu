#ifndef _GPU_MINER_KERNEL_CU_
#define _GPU_MINER_KERNEL_CU_
#include "CUDA_header.cuh"



void startTimer(unsigned *timer)
{
	CUT_SAFE_CALL( cutCreateTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( *timer));
}

double endTimer(char *info, unsigned *timer)
{
	hipDeviceSynchronize();
	CUT_SAFE_CALL( cutStopTimer( *timer));
	double result=cutGetTimerValue(*timer);
#ifdef PRINT_TIMER
	printf("***%s costs, %f, ms***\n", info, result);
#endif
	CUT_SAFE_CALL( cutDeleteTimer( *timer));
	return result/1000.0; //sec
}


#endif