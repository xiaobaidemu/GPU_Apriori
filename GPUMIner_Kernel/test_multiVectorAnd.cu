#ifndef _TEST_MULTI_VECTOR_AND_CU_
#define _TEST_MULTI_VECTOR_AND_CU_

#include "stdlib.h"

#include "CUDA_header.cu"
#include "gpu_multiVectorAnd.cu"
#include "GPUMiner_Kernel.h"
#include "gpu_vectorAnd.cu"
#include "gather.cu"

/*
void multi_AND_wS_woC(bytes_t* d_in1, 
					  cons t int* d_in1List,
					  const int in1ListLen, 
					  const bytes_t* d_in2, 
					  const int* d_in2List, const int in2ListLen, 
					  const int* d_in2ListBound,
					  const int segLen, 
					  bytes_t* d_out,
					  const unsigned int numBlock = 256, 
					  const unsigned int numThread = 256
					  )

*/

void test_multi_AND_wS_woC1()
{
	int segLen = 2;

	//cpu
	bytes_t h_in1[] = { 24, 56, 78, 97, 102, 562 };
	bytes_t h_in2[] = { 1, 2, 3, 4, 5, 6, 7, 8 };
	int in1Len = segLen*3;
	int in2Len = segLen*4;
	int h_in1List[] = { 0, 2 };
	int in1ListLen = 2;
	int h_in2List[] = { 0, 3, 1, 2, 3 };
	int in2ListLen = 5;
	int h_in2ListBound[] = { 2, 3 };	

	//gpu
	bytes_t *d_in1, *d_in2;
	copyToGPU<bytes_t>( &d_in1, h_in1, in1Len );
	copyToGPU<bytes_t>( &d_in2, h_in2, in2Len );
	int *d_in1List, *d_in2List, *d_in2ListBound;
	copyToGPU<int>( &d_in1List, h_in1List, in1ListLen );
	copyToGPU<int>( &d_in2List, h_in2List, in2ListLen );
	copyToGPU<int>( &d_in2ListBound, h_in2ListBound, in1ListLen );
	bytes_t* d_out;
	int outLen = segLen*in2ListLen;
	GPUMALLOC( (void**)&d_out, sizeof(bytes_t)*outLen );

	multi_AND_wS_woC( d_in1, d_in1List, in1ListLen, d_in2, d_in2List, in2ListLen, d_in2ListBound, segLen, d_out );

	bytes_t* h_out;
	copyFromGPU<bytes_t>( &h_out, d_out, outLen );
}


void test_multi_AND_wS_woC2()
{
	unsigned int segLen = 10000; //100K/8
	unsigned int numSeg = 10;
	unsigned int inLen = segLen*numSeg;

	int numTest = 1;

	//cpu
	bytes_t* h_in1 = (bytes_t*)malloc( sizeof(bytes_t)*inLen );
	bytes_t* h_in2 = (bytes_t*)malloc( sizeof(bytes_t)*inLen );
	randArray<bytes_t>( h_in1, inLen, 0, 65535, 777 );
	randArray<bytes_t>( h_in2, inLen, 0, 65535, 999 );
	int in1ListLen = 20;
	int* h_in1List = (int*)malloc( sizeof(int)*in1ListLen );
	int* h_in2ListBound = (int*)malloc( sizeof(int)*in1ListLen );
	randArrayByOrder<int>( h_in1List, in1ListLen, 0, numSeg - 1, 0 );
	randArray<int>( h_in2ListBound, in1ListLen, 60, 80, 1 ); //each segment in in1 matchs 'low' to 'high' segments in in2
	int in2ListLen = 0;
	for( int i = 0; i < in1ListLen; i++ )
		in2ListLen += h_in2ListBound[i];
	int* h_in2List = (int*)malloc( sizeof(int)*in2ListLen );
	int offset = 0;
	for( int i = 0; i < in1ListLen; i++ )
	{
		randArrayByOrder<int>( h_in2List + offset, h_in2ListBound[i], 0, numSeg - 1, 2 );
		offset += h_in2ListBound[i];
	}
	
	//gpu
	int outLen = in2ListLen*segLen;
	bytes_t* d_in1;
	bytes_t* d_in2;
	copyToGPU<bytes_t>( &d_in1, h_in1, inLen );
	copyToGPU<bytes_t>( &d_in2, h_in2, inLen );
	int *d_in1List, *d_in2List, *d_in2ListBound;
	copyToGPU<int>( &d_in1List, h_in1List, in1ListLen );
	copyToGPU<int>( &d_in2List, h_in2List, in2ListLen );
	copyToGPU<int>( &d_in2ListBound, h_in2ListBound, in1ListLen );
	bytes_t* d_out;
	GPUMALLOC( (void**)&d_out, sizeof(bytes_t)*outLen );

	//run multiAnd wSwoC on the GPU
	printTitle( "start multiAND wSwoC on the GPU..." );
	unsigned int gpuTimer1 = 0;
	startTimer( &gpuTimer1 );
	for( int i = 0; i < numTest; i++ )
	{
		multi_AND_wS_woC( d_in1, d_in1List, in1ListLen, d_in2, d_in2List, in2ListLen, d_in2ListBound, segLen, d_out );
	}	
	SYNC();
	endTimer( "GPU multiple", &gpuTimer1 );
	bytes_t* h_out;
	copyFromGPU<bytes_t>( &h_out, d_out, outLen );

	//run singleAnd wSwoC on the GPU
	arrayAndInit( in2ListLen, segLen );
	bytes_t* d_goldOut;
	GPUMALLOC( (void**)&d_goldOut, sizeof(bytes_t)*outLen );
	printTitle( "start singleAND wSwoC on the GPU..." );
	unsigned int gpuTimer2 = 0;
	startTimer( &gpuTimer2 );
	for( int i = 0; i < numTest; i++ )
	{
		offset = 0;
		int outOffset = 0;
		for( int i = 0; i < in1ListLen; i++ )
		{
			array_And_wS_woC( d_in1 + segLen*h_in1List[i], segLen, 
						   d_in2, h_in2List + offset, h_in2ListBound[i], 
						   d_goldOut + outOffset );
			outOffset += ( segLen*h_in2ListBound[i] );
			offset +=  h_in2ListBound[i];
		}
	}
	SYNC();
	endTimer( "GPU single", &gpuTimer2 );
	bytes_t* h_goldOut;
	copyFromGPU<bytes_t>( &h_goldOut, d_goldOut, outLen );

	checkResult( h_out, h_goldOut, outLen, "AND" );
}


void test_multi_AND_woS_wC2()
{
	unsigned int segLen = 10000; //100K/8
	unsigned int numSeg = 10;
	unsigned int inLen = segLen*numSeg;

	int numTest = 1;

	//cpu
	bytes_t* h_in1 = (bytes_t*)malloc( sizeof(bytes_t)*inLen );
	bytes_t* h_in2 = (bytes_t*)malloc( sizeof(bytes_t)*inLen );
	randArray<bytes_t>( h_in1, inLen, 0, 65535, 777 );
	randArray<bytes_t>( h_in2, inLen, 0, 65535, 999 );
	int in1ListLen = 20;
	int* h_in1List = (int*)malloc( sizeof(int)*in1ListLen );
	int* h_in2ListBound = (int*)malloc( sizeof(int)*in1ListLen );
	randArrayByOrder<int>( h_in1List, in1ListLen, 0, numSeg - 1, 0 );
	randArray<int>( h_in2ListBound, in1ListLen, 60, 80, 1 ); //each segment in in1 matchs 'low' to 'high' segments in in2
	int in2ListLen = 0;
	for( int i = 0; i < in1ListLen; i++ )
		in2ListLen += h_in2ListBound[i];
	int* h_in2List = (int*)malloc( sizeof(int)*in2ListLen );
	int offset = 0;
	for( int i = 0; i < in1ListLen; i++ )
	{
		randArrayByOrder<int>( h_in2List + offset, h_in2ListBound[i], 0, numSeg - 1, 2 );
		offset += h_in2ListBound[i];
	}

	printf( "in2ListLen = %d\n", in2ListLen );

	arrayAndInit( in2ListLen, segLen );
	
	//gpu
	int outLen = in2ListLen*segLen;
	bytes_t* d_in1;
	bytes_t* d_in2;
	copyToGPU<bytes_t>( &d_in1, h_in1, inLen );
	copyToGPU<bytes_t>( &d_in2, h_in2, inLen );
	int *d_in1List, *d_in2List, *d_in2ListBound;
	copyToGPU<int>( &d_in1List, h_in1List, in1ListLen );
	copyToGPU<int>( &d_in2List, h_in2List, in2ListLen );
	copyToGPU<int>( &d_in2ListBound, h_in2ListBound, in1ListLen );
	table_t* d_countOut;
	GPUMALLOC( (void**)&d_countOut, sizeof(table_t)*outLen );
	CUDA_SAFE_CALL( hipMemset( d_countOut, 0, sizeof(table_t)*outLen ) );

	//run multiAnd wSwoC on the GPU
	//printTitle( "start multiAND woSwC on the GPU..." );
	unsigned int gpuTimer1 = 0;
	startTimer( &gpuTimer1 );
		for( int i = 0; i < numTest; i++ )
		{
			multi_AND_woS_wC( d_in1, d_in1List, in1ListLen, d_in2, d_in2List, in2ListLen, d_in2ListBound, segLen, d_countOut );
		}	
		table_t* h_countOut;
		copyFromGPU<table_t>( &h_countOut, d_countOut, outLen );
		SYNC();
	endTimer( "GPU multiple", &gpuTimer1 );

	//run singleAnd wSwoC on the GPU
	table_t* d_goldCountOut;
	GPUMALLOC( (void**)&d_goldCountOut, sizeof(table_t)*outLen );
	//printTitle( "start singleAND woSwC on the GPU..." );
	table_t* h_goldCountOut;
	unsigned int gpuTimer2 = 0;
	startTimer( &gpuTimer2 );
	for( int i = 0; i < numTest; i++ )
	{
		offset = 0;
		int outOffset = 0;
		for( int i = 0; i < in1ListLen; i++ )
		{
			array_And_woS_wC( d_in1 + segLen*h_in1List[i], segLen, 
						   d_in2, h_in2List + offset, h_in2ListBound[i], 
						   d_goldCountOut + outOffset );
			outOffset += ( segLen*h_in2ListBound[i] );
			offset +=  h_in2ListBound[i];
		}		
		copyFromGPU<table_t>( &h_goldCountOut, d_goldCountOut, outLen );
	}
	SYNC();
	endTimer( "GPU single", &gpuTimer2 );

	//count on the CPU
	//run multiAnd wSwoC on the GPU
	/*
	bytes_t* d_out;
	table_t* d_cOut;
	GPUMALLOC( (void**)&d_cOut, sizeof(table_t)*outLen );
	GPUMALLOC( (void**)&d_out, sizeof(bytes_t)*outLen );
	//printTitle( "start multiAND wSwoC on the GPU..." );	
	table_t* cpu_out = (table_t*)malloc( sizeof(table_t)*outLen );
	unsigned int gpuTimer3 = 0;
	startTimer( &gpuTimer3 );
		for( int i = 0; i < numTest; i++ )
		{
			multi_AND_wS_woC( d_in1, d_in1List, in1ListLen, d_in2, d_in2List, in2ListLen, d_in2ListBound, segLen, d_out );
			
			//bytes_t* h_out;
			//copyFromGPU<bytes_t>( &h_out, d_out, outLen );
			//for( int i = 0; i < outLen; i++ )
			//	cpu_out[i] = h_byteTable[h_out[i]];
			

			gahter<table_t, table_t, bytes_t>( g_byteTable, outLen, d_cOut, d_out, outLen );			
			copyFromGPU<table_t>( &cpu_out, d_cOut, outLen );
		}	
		SYNC();
	endTimer( "GPU multiple", &gpuTimer3 );
	*/

	checkResult( h_countOut, h_goldCountOut, outLen, "AND" );
	//checkResult( h_goldCountOut, cpu_out, outLen, "AND" );
}

#endif