#include "hip/hip_runtime.h"
#ifndef _GPU_MINER_KERNEL_CUH_
#define _GPU_MINER_KERNEL_CUH_


//#include "gpu_multiVectorAnd.cu"
#include "gpu_vectorAndInit.cuh"
#include "CUDA_header.cuh"

////////////////////////////////////// Global Variable /////////////////////////
double copyTime = 0.0;
double countTime = 0.0;
double kernelTime = 0.0;

static __device__ __constant__ table_t c_byteTable[TABLE_SIZE];//�����ڴ治��hipMalloc
int* d_id2List;
bytes_t* d_buf;
table_t* g_byteTable;
int* d_in1List;
int* d_in2List;
int* d_in2ListBound;



__global__
void multi_AND_wS_woC_kernel(bytes_t* d_in1, 
							 const int* d_in1List,
							 const int in1ListLen, 
							 const bytes_t* d_in2, 
							 const int* d_in2List, const int in2ListLen, 
							 const int* d_in2ListBound,
							 const int segLen, 
							 bytes_t* d_out)
{
	//1. load the d_in1List, d_in2List and d_in2ListBound to the shared memory
	extern __shared__ int s_data[];
	int* s_in1List = s_data; //len = in1ListLen	
	int* s_in2ListBound = s_data + in1ListLen; //len = in1ListLen
	int* s_in2List = s_data + 2*in1ListLen; //len = in2ListLen
	//�о�������Բ�ʹ��threadIdx.x
	for( int i = threadIdx.x; i < in1ListLen; i += blockDim.x )
	{
		s_in1List[i] = d_in1List[i];
		s_in2ListBound[i] = d_in2ListBound[i];
	}
	for( int i = threadIdx.x; i < in2ListLen; i += blockDim.x )
	{
		//s_in2List��ÿһ���¼����һ��itemidֵ
		s_in2List[i] = d_in2List[i];
	}
	__syncthreads();

	//2.
	int outOffset = 0; //for output, also for get the d_segIn2List
	bytes_t* d_segIn1;//pointer to d_in1 for current segment
	int* d_segIn2List; //pointer to d_in2List for current segment
	int segIn2ListLen;
	int segOutLen = 0;
	const unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int delta = blockDim.x*gridDim.x;
	int idid2, offset, posInIn2;
	for( int id1 = 0; id1 < in1ListLen; id1++ )
	{
		//d_segIn1 ָ�����d_in1��ǰҪ�����Ƭ�Σ�����ǰ���ڵ�Ƭ�����е������01ֵ
		d_segIn1 = d_in1 + segLen*(s_in1List[id1]);
		//�������д��룬��ʾ��ѭ���������ڵ�Ϊid1���ӽڵ㣬������ͨ��GPUͬʱ����s_in2ListBound[id1]����Լ��200-400���ң��Ը��ӽڵ��������,ÿ���̴߳���16λ
		//���ݵ�������
		d_segIn2List = s_in2List + outOffset;		
		segIn2ListLen = s_in2ListBound[id1];

		segOutLen = segLen*segIn2ListLen;//segIn2ListLen��С��1000��ֵ
		//for each segment,ÿ���̴߳���16λ������������		
		//index��ȡֵ��Χ��0-1536��deltaΪ1536����ǰ��Ҫ����segOutLen��16λ���ݣ���ΪͬʱҪ����s_in2ListBound[id1]�Ը��ӽڵ�������㣬���Դ��������Ƚϸ���
		for( int i = index; i < segOutLen; i += delta )
		{
			//idid2 ��ʾ��ǰ����߳�������һ���ӽڵ�
			idid2 = i/segLen;
			//offset��d_segIn1��d_in2����ͬ���� offsetd��ȡֵ��Χ��0-(segLen-1),��֤�����������������
			offset = i - idid2*segLen;
			//posInIn2 ��ʾ�ڵ�ǰ���ڵ��£���Ҫ������ӽڵ��16λ���ݵ����������ֵҪ�Ͷ�Ӧ�ĸ��ڵ����Ӧ������d_segIn2List[idid2]��¼�����ӽڵ��Ӧ��item��id��ֵ
			//offsetҪ��֤������������ĸ��ӽڵ��ƫ��Ҫһ��
			posInIn2 = d_segIn2List[idid2]*segLen + offset;
			//d_out ����м�������������2*segLen*1500*15,(1500Ӧ�ú�1000���й�ϵ),15����˼�����level��ֵ������15��������Ҫ���levelPos�������
			//segOutLen < d_out��������
			//�����ȡd_out���Է��ڳ����洢����
			//���������ֵ����d_out�У�outOffset*segLen��Ϊ��ȷ����ǰ���ڵ���������ȫ���ӽڵ����м�����λ��
			d_out[i + outOffset*segLen] = d_segIn1[offset]&d_in2[posInIn2];
		}
		outOffset += (s_in2ListBound[id1]);
	}
}


__global__
void multi_AND_woS_wC_kernel(bytes_t* d_in1, 
							 const int* d_in1List,
							 const int in1ListLen, 
							 const bytes_t* d_in2, 
							 const int* d_in2List, const int in2ListLen, 
							 const int* d_in2ListBound,
							 const int segLen, 
							 table_t* d_countOut )
{
	//1. load the d_in1List, d_in2List and d_in2ListBound to the shared memory
	extern __shared__ int s_data[];
	int* s_in1List = s_data; //len = in1ListLen	
	int* s_in2ListBound = s_data + in1ListLen; //len = in1ListLen
	int* s_in2List = s_data + 2*in1ListLen; //len = in2ListLen

	for( int i = threadIdx.x; i < in1ListLen; i += blockDim.x )
	{
		s_in1List[i] = d_in1List[i];
		s_in2ListBound[i] = d_in2ListBound[i];
	}
	for( int i = threadIdx.x; i < in2ListLen; i += blockDim.x )
	{
		s_in2List[i] = d_in2List[i];
	}
	__syncthreads();

	//2.
	int outOffset = 0; //for output, also for get the d_segIn2List
	bytes_t* d_segIn1;//pointer to d_in1 for current segment
	int* d_segIn2List; //pointer to d_in2List for current segment
	int segIn2ListLen;
	int segOutLen = 0;
	const unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int delta = blockDim.x*gridDim.x;
	int idid2, offset, posInIn2;
	bytes_t out;
	for( int id1 = 0; id1 < in1ListLen; id1++ )
	{
		d_segIn1 = d_in1 + segLen*(s_in1List[id1]);
		d_segIn2List = s_in2List + outOffset;		
		segIn2ListLen = s_in2ListBound[id1];
		segOutLen = segLen*segIn2ListLen;
		//for each segment		
		for( int i = index; i < segOutLen; i += delta )
		{
			idid2 = i/segLen;
			//offset��d_segIn1��d_in2����ͬ���� offsetd��ȡֵ��Χ��0-(segLen-1),��֤�����������������
			offset = i - idid2*segLen;
			posInIn2 = d_segIn2List[idid2]*segLen + offset;
			out = d_segIn1[offset]&d_in2[posInIn2];
			//�����㵽����levelʱ��ֱ�Ӷ����ս����ֵ�������ж��ٸ�1
			d_countOut[i + outOffset*segLen] = c_byteTable[out];
			//d_out[i + outOffset*segLen] = d_segIn1[offset]&d_in2[posInIn2];
		}

		outOffset += (s_in2ListBound[id1]);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! GPU primitive for vector and operation for multiple d_in1 and multiple d_in2
//! @param d_in1
//! @param d_in1List list contains the id in d_in1
//! @param in1ListLen
//! @param d_in2
//! @param d_in2List list contains the id in d_in2
//! @param in2ListLen in2ListLen >= in1ListLen
//! @param d_in2ListBoud number of corresponding ids in d_in2 for each id in d_in1, len = in1ListLen
//! @param segLen size of each segment for AND operation
//! @param d_out, outLen = in2ListLen*segLen
////////////////////////////////////////////////////////////////////////////////
void multi_AND_wS_woC(bytes_t* d_in1, 
					  const int* d_in1List,
					  const int in1ListLen, 
					  const bytes_t* d_in2, 
					  const int* d_in2List, const int in2ListLen, 
					  const int* d_in2ListBound,
					  const int segLen, 
					  bytes_t* d_out,
					  const unsigned int numBlock = 8, 
					  const unsigned int numThread = 192
					  )
{
	//store the d_in1List, d_in2List and d_in2ListBound to the shared memory
	unsigned int sharedSize = sizeof(int)*( 2*in1ListLen + in2ListLen );
	multi_AND_wS_woC_kernel<<<numBlock, numThread, sharedSize>>>( d_in1, d_in1List, in1ListLen,
		d_in2, d_in2List, in2ListLen, d_in2ListBound, segLen, d_out);
	CUT_CHECK_ERROR( "multi_AND_wS_woC_kernel" );
	SYNC();
}


void multi_AND_wS_woC_hostList(bytes_t* d_in1, 
							   const int* h_in1List,
							   const int in1ListLen, 
							   const bytes_t* d_in2, 
							   const int* h_in2List, const int in2ListLen, 
							   const int* h_in2ListBound,
							   const int segLen, 
							   bytes_t* d_out,
							   const unsigned int numBlock = 8, 
							   const unsigned int numThread = 192
							   )
{
	unsigned int copyTimer = 0;
	startTimer( &copyTimer );
	TOGPU( d_in1List, h_in1List, sizeof(int)*in1ListLen );
	TOGPU( d_in2ListBound, h_in2ListBound, sizeof(int)*in1ListLen );
	TOGPU( d_in2List, h_in2List, sizeof(int)*in2ListLen );
	copyTime += endTimer( "", &copyTimer );

	unsigned int kernelTimer = 0;
	startTimer( &kernelTimer );
	multi_AND_wS_woC(d_in1, d_in1List,
		in1ListLen, 
		d_in2, 
		d_in2List, in2ListLen, 
		d_in2ListBound,
		segLen, 
		d_out);
	kernelTime += endTimer( "", &kernelTimer );
}

//!!!!!
void multi_AND_woS_wC(bytes_t* d_in1, 
					  const int* d_in1List,
					  const int in1ListLen, 
					  const bytes_t* d_in2, 
					  const int* d_in2List, const int in2ListLen /**/, 
					  const int* d_in2ListBound,
					  const int segLen, 
					  table_t* d_countOut,
					  const unsigned int numBlock = 8, 
					  const unsigned int numThread = 192
					  )
{

	//store the d_in1List, d_in2List and d_in2ListBound to the shared memory
	unsigned int sharedSize = sizeof(int)*( 2*in1ListLen + in2ListLen );

	multi_AND_woS_wC_kernel<<<numBlock, numThread, sharedSize>>>( d_in1, d_in1List, in1ListLen,
		d_in2, d_in2List, in2ListLen, d_in2ListBound, segLen, d_countOut);
	CUT_CHECK_ERROR( "multi_AND_wS_woC_kernel" );
	SYNC();			

}


void multi_AND_woS_wC_hostList(bytes_t* d_in1, 
							   const int* h_in1List,
							   const int in1ListLen, 
							   const bytes_t* d_in2, 
							   const int* h_in2List, const int in2ListLen /**/, 
							   const int* h_in2ListBound,
							   const int segLen, 
							   table_t* d_countOut,
							   const unsigned int numBlock = 8, 
							   const unsigned int numThread = 192
							   )
{
	unsigned int copyTimer = 0;
	startTimer( &copyTimer );
	TOGPU( d_in1List, h_in1List, sizeof(int)*in1ListLen );
	TOGPU( d_in2ListBound, h_in2ListBound, sizeof(int)*in1ListLen );
	TOGPU( d_in2List, h_in2List, sizeof(int)*in2ListLen );
	copyTime += endTimer( "", &copyTimer );
	unsigned int kernelTimer = 0;
	startTimer( &kernelTimer );
	multi_AND_woS_wC(d_in1, 
		d_in1List,
		in1ListLen, 
		d_in2, 
		d_in2List, in2ListLen /**/, 
		d_in2ListBound,
		segLen, 
		d_countOut
		);
	kernelTime += endTimer( "", &kernelTimer );
}

bytes_t* d_bitmap;
bytes_t* d_midRes;
table_t* d_byteTable;
int itemSize;
int numTran;
int numTranInByte;
bytes_t* d_multiOut;
bytes_t* h_multiOut;
table_t* d_multiCountOut;
table_t* h_multiCountOut;
int* d_countBuf; 
#define CPU_COUNT
//#define GPU_MINER_DEBUG

template<class T>
void copyFromGPU( T** h_out, T* d_in, unsigned int len )
{
	CPUMALLOC( (void**)&(*h_out), sizeof(T)*len );
	FROMGPU( (*h_out), d_in, sizeof(T)*len );
}

template<class T>
void copyToGPU( T** d_out, T* h_in, unsigned int len )
{
	GPUMALLOC( (void**)&(*d_out), sizeof(T)*len );
	TOGPU( (*d_out), h_in, sizeof(T)*len );
}


//////////////////////////////////////// The basic GPU functions //////////////////

extern "C"
void GPUInit( int argc, char** argv )
{
	CUT_DEVICE_INIT( argc, argv );
}

extern "C"
void gpuMalloc( void** gpu_data, unsigned int sizeInByte )
{
	GPUMALLOC( gpu_data, sizeInByte );
}

extern "C"
void copyCPUToGPU( void* cpu_data, void* gpu_data, unsigned int sizeInByte )
{
	TOGPU( gpu_data, cpu_data, sizeInByte );
}

extern "C"
void copyCPUToGPUConstant( void* cpu_data, void* gpu_data, unsigned int sizeInByte )
{
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( gpu_data), cpu_data, sizeInByte ) );
}

extern "C"
void copyGPUToCPU( void* gpu_data, void* cpu_data, unsigned int sizeInByte )
{
	FROMGPU( cpu_data, gpu_data, sizeInByte );
}

extern "C"
void GPUFree( void* gpu_data )
{
	CUDA_SAFE_CALL( hipFree( gpu_data ) );
}

/////////////////////////////////////////////// GPUMiner ////////////////////////

extern "C"
void GPUMiner_Free()
{
	GPUFREE( d_bitmap );
	GPUFREE( d_midRes );
	GPUFREE( d_byteTable );
	GPUFREE( d_multiCountOut );
	
#ifdef CPU_COUNT
	CPUFREE( h_multiCountOut );
#endif
}



extern "C"
void arrayAndInit( const unsigned int maxListLen, const unsigned int numTranInByte )
{
	//����ʹ�ó����ڴ����������Ч�ʺ�ʱ�䣬��Ҫ��h_byteTable����ŵ�GPU�ĳ����ڴ��в�ѯ��
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( c_byteTable), h_byteTable, sizeof(table_t)*TABLE_SIZE ) );	
	GPUMALLOC( (void**)&d_id2List, sizeof(int)*maxListLen );	//maxListLen ��ʼ�������Ƶ��1�����

	GPUMALLOC( (void**)&d_in1List, sizeof(int)*NUM_MAX_NODE_PER_CALL );
	GPUMALLOC( (void**)&d_in2List, sizeof(int)*NUM_MAX_NODE_PER_CALL );
	GPUMALLOC( (void**)&d_in2ListBound, sizeof(int)*NUM_MAX_NODE_PER_CALL );

#ifdef USE_GATHER_FOR_COUNT
	GPUMALLOC( (void**)&g_byteTable, sizeof(table_t)*TABLE_SIZE );
	TOGPU( g_byteTable, h_byteTable, sizeof(table_t)*TABLE_SIZE  );
	GPUMALLOC( (void**)&d_multiOut, sizeof(bytes_t)*numTranInByte*NUM_MAX_NODE_PER_CALL );
#endif

#ifdef USE_CONSTANT_BUF
	CPUMALLOC( (void**)&hh_in, sizeof(bytes_t)*IN_SIZE );
#endif
#ifdef USE_COPY
	GPUMALLOC( (void**)&d_buf, sizeof(bytes_t)*maxListLen*numTranInByte );
#endif
}
//��ʼ��GPU�������ھ�
/*
* h_numTran:��ǰ���������
* h_itemSize:Ƶ��1��ĸ���
*/
extern "C"
void initGPUMiner( bytes_t* h_matrix, const unsigned int h_itemSize, const unsigned int h_numTran )
{
	numTran = h_numTran;
	numTranInByte = (int)ceil((float)numTran/NUM_BIT_IN_BYTE);//NUM_BIT_IN_BYTE = 16������ȡ����ΪʲôҪ����һ���ֽ�����16λ
	itemSize = h_itemSize;	

	unsigned int memTimer = 0;
	startTimer( &memTimer );
	//����GPU���ڴ棬����Ĵ�СΪ������Ϊ������ռ�ֽ���*Ƶ��1�����
	GPUMALLOC( (void**)&d_bitmap, sizeof(bytes_t)*numTranInByte*itemSize );//��Ϊ�ոն����2�������������sizeof(bytes_t),����ΪʲôҪ��ô��
	//����hipMemcpy��CPU�е�h_matrix���Ƶ�d_bitmap
	TOGPU( d_bitmap, h_matrix, sizeof(bytes_t)*numTranInByte*itemSize);
	//Ϊ�м�������GPU�ڴ�ռ�midRes
	//GPUMALLOC( (void**)&d_midRes, sizeof(bytes_t)*numTranInByte*itemSize*NUM_MAX_LEVEL );
	//GPUInit<bytes_t>( d_midRes, numTranInByte*itemSize, 0 );
	GPUMALLOC( (void**)&d_midRes, sizeof(bytes_t)*numTranInByte*NUM_MAX_NODE_PER_CALL*NUM_MAX_LEVEL );
	CUDA_SAFE_CALL( hipMemset(d_midRes, 0, sizeof(bytes_t)*numTranInByte*NUM_MAX_NODE_PER_CALL*NUM_MAX_LEVEL) );

	//initialize for the andVector��ʼ����Ϊ������׼��
	arrayAndInit( itemSize, numTranInByte);
	GPUMALLOC( (void**)&d_multiCountOut, sizeof(table_t)*numTranInByte*NUM_MAX_NODE_PER_CALL ); ///////////////!!!!!!! can be deleted

#ifdef CPU_COUNT
	GPUMALLOC( (void**)&d_multiOut, sizeof(bytes_t)*numTranInByte*NUM_MAX_NODE_PER_CALL );
	CPUMALLOC( (void**)&h_multiOut, sizeof(bytes_t)*numTranInByte*NUM_MAX_NODE_PER_CALL );
	CPUMALLOC( (void**)&h_multiCountOut, sizeof(table_t)*numTranInByte*NUM_MAX_NODE_PER_CALL );
#else
	GPUMALLOC( (void**)&d_countBuf, sizeof(int)*NUM_MAX_NODE_PER_CALL );
#endif
	copyTime = endTimer("", &memTimer);
}

int levelPos[NUM_MAX_LEVEL] = {0};

//update the level size
inline void updateLevelSize( int levelIdx, const int levelSize )
{
	levelPos[levelIdx] = levelPos[levelIdx - 1] + levelSize;
}

//get the starting address of offset in levelIdx
//ȡ��levelIdx��ʼ��ַ��ƫ��
inline int getLevelPos( const int levelIdx, const int offset )
{
	return levelPos[levelIdx] + offset*numTranInByte;
}


//left: int midRltBeginPos, int *parentList
//if midRltBeginPos = -1, parentList->matrix
//if midRltBeginPos != -1, midRest's offset, last level
//right: 
//itemIdList: matrix
//int *itemLenList, int pairNum
//midRltStoreLevel: if no counting, store
//|paraentList| and |itemLenList| = pairNum
//|itemIdList| = itemIdListLen
extern "C"
void gpuBoundVectorAnd( const int midRltBeginPos, const int *parentList, const int *itemLenList, const int pairNum,
						const int *itemIdList, const int itemIdListLen, const int midRltStoreLevel, 
						const bool countSup, int *supList)
{
	bytes_t* d_in1;
	bytes_t* d_in2;

	if( midRltBeginPos == -1 )
	//if(!countSup)
	{
		d_in1 = d_bitmap; //from the bitmap
	}
	else
	{
		//��Ҫ���м�����ʼ
		//��level=3ʱ��d_in1=d_midRes+gtLevelPos(1,0)��max_level=15,d_in1ָ���м�����ʼ��λ��
		/*
		inline int getLevelPos( const int levelIdx, const int offset )
		{
			return levelPos[levelIdx] + offset*numTranInByte;
		}
		*/
		//
		d_in1 = d_midRes + getLevelPos( midRltStoreLevel - 2, midRltBeginPos );//from the midRes	
	}

	d_in2 = d_bitmap;

#ifdef GPU_MINER_DEBUG
	bytes_t* h_in1;
	bytes_t* h_in2;
	copyFromGPU<bytes_t>( &h_in1, d_in1, numTranInByte*pairNum );
	copyFromGPU<bytes_t>( &h_in2, d_in2, numTranInByte*itemSize );
	int a = 1;
#endif

	if( countSup )//no store, do counting
	{
         //unsigned int kernelTimer = 0;
		 //startTimer( &kernelTimer );
		 multi_AND_woS_wC_hostList(d_in1, parentList, pairNum, 
							  d_in2, 
							  itemIdList, itemIdListLen, 
							  itemLenList,
							  numTranInByte, 
							  d_multiCountOut );
		 //kernelTime += endTimer( "", &kernelTimer );

#ifdef CPU_COUNT
		unsigned int copyTimer = 0;
		startTimer( &copyTimer );
		//��d_multiCountOut��GPU�п�����CPU��
			FROMGPU( h_multiCountOut, d_multiCountOut, sizeof(table_t)*numTranInByte*itemIdListLen );
		copyTime += endTimer("", &copyTimer);
		
		unsigned int countTimer = 0;
		startTimer( &countTimer );
			for( int i = 0; i < itemIdListLen; i++ )
			{
				int sum = 0;
				for( int j = 0; j < numTranInByte; j++ )
				{
					sum += h_multiCountOut[i*numTranInByte + j];
				}
				supList[i] = sum;
			}
		countTime += endTimer( "", &countTimer );		
#else
		unsigned int countTimer = 0;
		startTimer( &countTimer );
			fixlenSum_v2<table_t, int>( d_multiCountOut, d_countBuf, numTranInByte, itemIdListLen, supList ); 		 
		countTime += endTimer( "", &countTimer );
#endif
		
	}
	else //store, no counting
	{
		//��midRltStoreLevel = 2
		bytes_t* d_out = d_midRes + levelPos[midRltStoreLevel - 1];

		multi_AND_wS_woC_hostList( d_in1, parentList, pairNum, 
					  d_in2, 
					  itemIdList, itemIdListLen, 
					  itemLenList,
					  numTranInByte, 
					  d_out
					  );
		//�м���ռ����itemIdListLen*numTranInByte��˫�ֽ�
		/*
		inline void updateLevelSize( int levelIdx, const int levelSize )
		{
			levelPos[levelIdx] = levelPos[levelIdx - 1] + levelSize;
		}
		*/
		//updateLevelSize ����levelPos(��ǰ�������)��ֵ����ǰһ����ȼ���itemIdListLen*numTranInByte
		updateLevelSize( midRltStoreLevel, itemIdListLen*numTranInByte );//���µ�����һ���м�����ʼ��λ��
	}

#ifdef GPU_MINER_DEBUG
		bytes_t* h_midRes;
		copyFromGPU<bytes_t>( &h_midRes, d_midRes, numTranInByte*NUM_MAX_NODE_PER_CALL*NUM_MAX_LEVEL );
		a = 1;
#endif
}



extern "C"
double getCopyTime()
{
	double timer = copyTime;
	copyTime = 0.0;
	return timer;
}

extern "C"
double getCountTime()
{
	double timer = countTime;
	countTime = 0.0;
	return timer;
}

extern "C"
double getKernelTime()
{
	double timer = kernelTime;
	kernelTime = 0.0;
	return timer;
}

#endif